#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <>
#include <gtest/gtest.h>
#include <tuple>
#include <chrono>
#include <functional>
#include <chrono>
#include <string>

__global__ 
void vec_add(float *A, float *B, float *C, long long int n)
{
    long long int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n)
        C[i] = A[i] + B[i];
}

void create_vec(float *A,long long  int n)
{
    // A = new float[n];
    srand (static_cast <unsigned> (time(0)));
    for(long long int i=0;i<n;i++)
        A[i] = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/1000000.0));
        // A[i] = i;
}

void checker(float *c1, float *c2, long long int n)
{
    for(long long int i=0;i<n;i++)
    {
        ASSERT_EQ(c1[i], c2[i]);
    }
}



void gpu_add (float *A, float *B, float *C, long long int n)
{
    long long int size = n*sizeof(float);

    float *d_a, *d_b, *d_c;
    // float *C;

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, size, hipMemcpyHostToDevice);

    vec_add<<<std::ceil(n/256.0),256>>>(d_a, d_b, d_c, n);     

    hipMemcpy(C, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);    
    hipFree(d_c);
    
}

void cpu_add (float *A, float *B, float *C, long long int n)
{
    for(long long int i=0;i<n;i++)
    {
        C[i] = A[i] + B[i];
    }
}

template <typename T>
void print_array( T* array,long long  int n)
{
    for(long long int i=0;i<n;i++)
        std::cout<<array[i]<<" ";
    std::cout<<std::endl;
}

void get_time(std::function<void(float*, float*, float*, long long int)> func, float* A, float* B, float* C, long long int n) 
{

    auto start = std::chrono::high_resolution_clock::now();
    func(A,B,C,n);
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    std::cout<< "Time taken by function: "<< duration.count() << " microseconds" << std::endl;
}

void initialize(long long int n)
{
    float *A;
    A = (float *)malloc(n*sizeof(float));
    float *B;
    B = (float *)malloc(n*sizeof(float));

    float *C1;
    C1 = (float *)malloc(n*sizeof(float));
    float *C2;
    C2 = (float *)malloc(n*sizeof(float));
    create_vec(A,n);
    create_vec(B,n);

    // bool check = true;

    // print_array<float>(A, n);
    // print_array<float>(A, n);

    // std::function
    std::function<void(float*, float*, float*, long long int)>cpufun = cpu_add;
    std::function<void(float*, float*, float*, long long int)>gpufun = gpu_add;

    get_time(cpufun,A,B,C1,n);
    get_time(gpufun,A,B,C2,n);

    // cpu_add(A,B, C1,n);
    // gpu_add(A,B, C2,n);

    // print_array<float>(C1, n);
    // print_array<float>(C2, n);

    checker(C1, C2, n);

    // delete A, B, C1, C2;
    delete A;
}

int main(int argc, char** argv) {

    long long int n = std::stoi(argv[1]);

    initialize(n);
    return 0;

}
