// This program computes matrix multiplication using shared memory tiling
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <chrono>
using std::cout;
using std::generate;
using std::vector;

// Pull out matrix and shared memory tile size 
const int N = 1 << 10;
const int SHMEM_SIZE = 1 << 8;

__global__ void matrixMul(const int *a, const int *b, int *c, int N) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterate over row, and down column
  c[row * N + col] = 0;
  for (int k = 0; k < N; k++) {
    // Accumulate results for a single element
    c[row * N + col] += a[row * N + k] * b[k * N + col];
  }
}

__global__ void tiled_matrixMul(const int *a, const int *b, int *c) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Statically allocated shared memory
  __shared__ int s_a[SHMEM_SIZE];
  __shared__ int s_b[SHMEM_SIZE];

  // Accumulate in temporary variable
  int tmp = 0;

  // Sweep tile across matrix
  for (int i = 0; i < N; i += blockDim.x) {
    // Load in elements for this tile
    s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * N + i + threadIdx.x];
    s_b[threadIdx.y * blockDim.x + threadIdx.x] = b[i * N + threadIdx.y * N + col];

    // Wait for both tiles to be loaded in before doing computation
    __syncthreads();

    // Do matrix multiplication on the small matrix
    for (int j = 0; j < blockDim.x; j++) {
      tmp +=
          s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
    }

    // Wait for all threads to finish using current tiles before loading in new
    // ones
    __syncthreads();
  }

  // Write back results
  c[row * N + col] = tmp;
}

// Check result on the CPU
void verify_result(vector<int> &a, vector<int> &b, vector<int> &c) {
  // For every row...
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      int tmp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        tmp += a[i * N + k] * b[k * N + j];
      }

      // Check against the CPU result
      assert(tmp == c[i * N + j]);
    }
  }
}

int main() {
  // Size (in bytes) of matrix
  size_t bytes = N * N * sizeof(int);

  // Host vectors
  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);
  vector<int> h_d(N * N);

  // Initialize matrices
  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  // Allocate device memory
  int *d_a, *d_b, *d_c, *d_d;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  hipMalloc(&d_d, bytes);

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 16;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = N / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  auto start = std::chrono::high_resolution_clock::now();
  tiled_matrixMul<<<blocks, threads>>>(d_a, d_b, d_c);
  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);
  auto stop = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
  std::cout<< "Time taken by function 1 (tiled_matrixMul): "<< duration.count() << " microseconds" << std::endl;
  
  start = std::chrono::high_resolution_clock::now();
  matrixMul<<<blocks, threads>>>(d_a, d_b, d_d,N);
  hipMemcpy(h_d.data(), d_d, bytes, hipMemcpyDeviceToHost);
  stop = std::chrono::high_resolution_clock::now();
  duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
  std::cout<< "Time taken by function 2 (matrixMul): "<< duration.count() << " microseconds" << std::endl;
  
  // Copy back to the host





  // Check result
  verify_result(h_a, h_b, h_c);
  verify_result(h_a, h_b, h_d);

  cout << "COMPLETED SUCCESSFULLY\n";

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_d);

  return 0;
}