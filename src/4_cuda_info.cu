#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <>
#include <gtest/gtest.h>
#include <tuple>
#include <chrono>
#include <functional>
#include <chrono>
#include <string>

int main()
{
    int count;
    hipGetDeviceCount(&count);
    std::cout << "Device count: " << count << std::endl;

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    std::cout << "threads per block: " << props.maxThreadsPerBlock << std::endl;
    std::cout << "multi-processor count: " << props.multiProcessorCount << std::endl;
    std::cout << "clock rate: " << props.clockRate << std::endl;

    std::cout << "max thread dim 0: " << props.maxThreadsDim[0] << std::endl;
    std::cout << "max thread dim 1: " << props.maxThreadsDim[1] << std::endl;
    std::cout << "max thread dim 2: " << props.maxThreadsDim[2] << std::endl;

    std::cout << "max grid dim 0: " << props.maxGridSize[0] << std::endl;
    std::cout << "max grid dim 1: " << props.maxGridSize[1] << std::endl;
    std::cout << "max grid dim 2: " << props.maxGridSize[2] << std::endl;

    return 0;

}